#include "hip/hip_runtime.h"
/* ########################################################################################################################
## Organization         : The University of Arizona
##                      :
## File name            : GaB.cu
## Language             : C (ANSI)
## Short description    : Gallager-B Hard decision Bit-Flipping algorithm
##                      :
##                      :
##                      :
## History              : Modified 12/04/2023
## ########################################################################################################################*/
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <stdio.h>
#include <unistd.h>
#include "GaB.h"

//#####################################################################################################
// Update VN to CN message array [Includes Multi Codeword Processing]
__global__ void DataPassGB(int *VtoC, int *CtoV, int *Receivedword, int *Interleaver,int ColumnDegree,int N,int NbBranch, int iter, int numWords)
{
	int t,numB,n,buf;
	int Global;
	// Calc relative global memory index where n spans multiple concatenated arrays for multiple words
    n = threadIdx.x + blockIdx.x*blockDim.x;
    // Spaced position in interleaver matrix where modulo operation allows for multi word wrapping
    numB = (ColumnDegree * n) % NbBranch;
    // Find which CW in concatenated array the thread is associated and calculate the offset for the concatenated array
    int CW_offset = (n / N) * NbBranch;

                    // DEBUG CODE
            //if (n < N)
             //  printf("CW offset = %d \n",CW_offset);



    // Conditional is boundary check
    if (n < N*numWords) {
        if (iter == 0) {
            for (t=0;t<ColumnDegree;t++)     
               VtoC[Interleaver[numB+t] + CW_offset]=Receivedword[n];
        }
        else {
		    //Global=(Amplitude)*(1-2*ReceivedSymbol[n]);
		    Global=(1-2*Receivedword[n]); 
		    //Global=(1-2*(Decide[n] + Receivedword[n])); //Decide[n]^Receivedword[n];
		    for (t=0;t<ColumnDegree;t++) 
                Global+=(-2)*CtoV[Interleaver[numB+t] + CW_offset]+1;

		    for (t=0;t<ColumnDegree;t++) {
		        buf=Global-((-2)*CtoV[Interleaver[numB+t] + CW_offset]+1);
		        if (buf<0)  
                    VtoC[Interleaver[numB+t] + CW_offset]= 1; //else VtoC[Interleaver[numB+t]]= 1;
		        else if (buf>0) 
                    VtoC[Interleaver[numB+t] + CW_offset]= 0; //else VtoC[Interleaver[numB+t]]= 1;
		        else  
                    VtoC[Interleaver[numB+t] + CW_offset]=Receivedword[n];
		    }
        }
    }
}

//##################################################################################################
// Update the CN to VN message array [Includes Multi Codeword Processing]
// Naive implemenation
__global__ void CheckPassGB(int *CtoV,int *VtoC,int M,int NbBranch,int RowDegree, int numWords)
{
    int t,numB=0,m,signe;
    // Calc relative global memory index where m spans multiple concatenated message arrays
    m = threadIdx.x + blockIdx.x*blockDim.x;
    // Calculate strided position for message arrays
    numB = (RowDegree * m) % NbBranch;
    // Find CW offset in concatenated array 
    int CW_offset = (m / M) * NbBranch;

    // Conditional is boundary check
    if (m < M*numWords) {
        signe=0;
        for (t=0;t<RowDegree;t++) {
            signe^=VtoC[numB+t + CW_offset];
        }
        for (t=0;t<RowDegree;t++) {     
            CtoV[numB+t + CW_offset]=signe^VtoC[numB+t + CW_offset];
        }
    }
}

// Faux memory access simulation
/*
__global__ void CheckPassGB(int *CtoV,int *VtoC,int M,int NbBranch,int RowDegree, int numWords)
{
    int t,numB=0,m,signe;
    // Calc relative global memory index where m spans multiple concatenated arrays for multiple words
    m = threadIdx.x + blockIdx.x*blockDim.x;
    // Calculate strided position for message arrays
    numB = (RowDegree * m) % NbBranch;
    // Find which CW in concatenated array the thread is associated and calculate the offset for the concatenated array
    int CW_offset = (m / M) * NbBranch;
    int offset2 = m % M;

    // Conditional is boundary check
    if (m < M*numWords) {
        signe=0;
        for (t=0;t<RowDegree;t++) {
            //signe^=VtoC[numB+t + CW_offset];
            signe^=VtoC[offset2 + t*M + CW_offset];

        }

        for (t=0;t<RowDegree;t++) {     
            //CtoV[numB+t + CW_offset]=signe^VtoC[numB+t + CW_offset];
            CtoV[offset2 + t*M + CW_offset]=signe^VtoC[offset2 + t*M + CW_offset];
        }
            
    }
}
*/

// Reduction based implemenation
/*
__global__ void CheckPassGB(int *CtoV,int *VtoC,int M,int NbBranch,int RowDegree, int numWords)
{
    // Calc relative global memory index where m spans multiple concatenated message arrays for multiple words
    int m = threadIdx.x + blockIdx.x*blockDim.x;
    int tid = threadIdx.x;

    // shared memory declaration for a copy of the VtoC message array (blocksize*row degree)
    __shared__ int sh_VtoC[128];
    // shared memory declaration for signe (blocksize* 1/2 * row degree)
    __shared__ int sh_signe[128];

    // Conditional is boundary check
    if (m < NbBranch*numWords) {

        // pull in from global to shared memory and sync threads before subsequent computations occur
        sh_VtoC[tid] = VtoC[m];
        __syncthreads();
        // make copy and sync threads
        sh_signe[tid] = sh_VtoC[tid];
        __syncthreads();

        // Reduction to single signe value for each CN
        // Reduction loop set up to limit thread control divergence
        for (int boundary = blockDim.x; boundary > blockDim.x/RowDegree; boundary = boundary/2) {
            if (tid < boundary/2) {
                int tmp = sh_signe[tid*2] ^ sh_signe[tid*2 + 1];
                __syncthreads();
                sh_signe[tid] = tmp;
                __syncthreads();
            }
        }
        // Sync threads before writing to global memory then construct
        __syncthreads();
        CtoV[m] = sh_signe[tid / RowDegree] ^ sh_VtoC[tid];
    
    }
}
*/

//#####################################################################################################
//  Update the VN's [Includes Multi Codeword Processing]
__global__ void APP_GB(int *Decide,int *CtoV,int *Receivedword,int *Interleaver,int ColumnDegree,int N,int M,int NbBranch, int numWords)
{
   	int t,numB,n,buf;
	int Global;
    // Calc relative global memory index where n spans multiple concatenated arrays for multiple words
    n = threadIdx.x + blockIdx.x*blockDim.x;
	// Spaced position in interleaver matrix where modulo operation allows for multi word wrapping
    numB = (ColumnDegree * n) % NbBranch;
    // Find which CW in concatenated array the thread is associated and calculate the offset for the concatenated array
    int CW_offset = (n / N) * NbBranch;



    
    // Conditional is boundary check
    if (n < N*numWords) {
		Global=(1-2*Receivedword[n]);



		for (t=0;t<ColumnDegree;t++) 
            Global+=(-2)*CtoV[Interleaver[numB+t] + CW_offset]+1;


        if(Global>0) 
            Decide[n]= 0;
        else if (Global<0) 
            Decide[n]= 1;
        else  
            Decide[n]=Receivedword[n];
    }


}

//#####################################################################################################
// Calculate Syndrome; determine if corrected word is valid codeword

// This kernel is from the Single CW per kernel prototype 
/*
__global__ void ComputeSyndrome(int *Decide,int *Mat,int *RowDegree,int M, int *Dev_Syndrome, int numWords)
{
	int Synd,k,l;
    //This needs reduction function 
    __shared__ int sh_Synd[648];
    
     int n = threadIdx.x + blockIdx.x*blockDim.x;
     int thd_id = threadIdx.x;

     if(n ==0 ) *Dev_Syndrome = 1;
     
     for (l=0;l<RowDegree[n];l++)Synd=Synd^Decide[Mat[n*8 + l]];    

     if (n < M) sh_Synd[thd_id] = Synd; 
     __syncthreads();
     
    //Reduce to a single value 
    for(int stride = blockDim.x/2 ; stride > 0; stride = stride/2) {
     sh_Synd[thd_id] = sh_Synd[thd_id] | sh_Synd[thd_id + stride];
     __syncthreads();
     }
    
     if (thd_id == 0 ) atomicMin(Dev_Syndrome, (1 - sh_Synd[0])); 

}
*/

// This is a temp kernel w/o optimzation in mind
/*
__global__ void ComputeSyndrome(int *Decide,int *Mat,int RowDegree,int M, int *Dev_Syndrome, int numWords)
{
	int Synd,k,l,i;

    // Single thread per CW Syndrome calculation
    i = threadIdx.x * 1296;
    
	for (k=0;k<M;k++) {
		Synd=0;
		for (l=0;l<RowDegree;l++) {
            Synd=Synd^Decide[Mat[k*RowDegree+l] + i];
            //printf("Kernel Internal Synd =  %d  \n",Synd);

        }
        
        if (Synd == 1)
            break;

    }

    // Update Syndrome tracker array; each entry in array is assigned to single CW syndrome result
    Dev_Syndrome[threadIdx.x] = 1-Synd;
}
*/

// This kernel is for multiple codeword 
__global__ void ComputeSyndrome(int *Decide,int *Mat,int RowDegree,int M,
                                           int *Dev_Syndrome, int numWords)
{
	int Synd = 0,k,l;
    //Shared memory to utilize reduction operation  
    __shared__ int sh_Synd[512];
    //Pointer to hold the starting point of Decide array of operating codeword
    //int *Decide_skid;

     //Global thread Index. Shall be in the range of 0 to num_codeword * 2048
     int n = threadIdx.x + blockIdx.x*blockDim.x;
     //Thread Index at thread block level 
     int thd_id = threadIdx.x;
    
     //Initialize tshared memory to 0 and wait for all threads to complete 
     sh_Synd[thd_id] = 0; __syncthreads();
   
     int cw_operated = n/1024; //Find the CW on which thd block is operating 

     //Initialize the Global memory Dev_Syndrome to 1 for each codeword 
     if(n %1024 == 0 ) Dev_Syndrome[cw_operated] = 1; __syncthreads();
          
         
     int idx = (n %1024); //Find the bit location on the operating codeword 
     int vld_idx = idx < M; //Qual to check the thd is in H Mat Row range[0-647]

     //Find the Decide location for the operating codeword 
     __syncthreads();
   
     //Check bit level syndrome     
     //if (vld_idx) {
    	 for (l=0;l<RowDegree;l++)Synd=Synd^Decide[Mat[idx*8 + l] + (cw_operated * 1296) ];    
     	 if (vld_idx) sh_Synd[thd_id] = Synd; 
     //}
     __syncthreads();
     
    //Reduce to a single value 
    for(int stride = blockDim.x/2 ; stride > 0; stride = stride/2) {
     sh_Synd[thd_id] = sh_Synd[thd_id] | sh_Synd[thd_id + stride];
     __syncthreads();
     } 
     //Write back to Global memory
     if (thd_id == 0 ) atomicMin(&Dev_Syndrome[cw_operated],(1 - sh_Synd[0])); 
     // if (vld_idx ) atomicMin(Dev_Syndrome+ (cw_operated << 2),(1 - sh_Synd[thd_id]));
     // printf ("\n %x",  Dev_Syndrome+ (cw_operated << 2));
     __syncthreads();

}
